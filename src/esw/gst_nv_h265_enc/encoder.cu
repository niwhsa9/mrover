#include "gst_nv_h265_enc.hpp"

#include <hip/hip_runtime.h>

#include <cassert>

hipCtx_t CUDA_CONTEXT = nullptr;

void init_cuda() {
    auto result = hipInit(0);
    assert(result == hipSuccess);
}

