#include "hip/hip_runtime.h"
// Be careful what you include in this file, it is compiled with nvcc (NVIDIA CUDA compiler)
// For example OpenCV and lie includes cause problems

#include "zed_wrapper.hpp"

#include <sl/Camera.hpp>

namespace mrover {

    // Optimal for the Jetson Xavier NX - this is max threads per block and each block has a max of 2048 threads
    constexpr uint BLOCK_SIZE = 1024;

    /**
     * @brief Runs on the GPU, interleaving the XYZ and BGRA buffers into a single buffer of #Point structs.
     */
    __global__ void fillPointCloudMessageKernel(sl::float4* xyzGpuPtr, sl::uchar4* bgraGpuPtr, Point* pcGpuPtr, size_t size) {
        // This function is invoked once per element at index #i in the point cloud
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= size) return;

        pcGpuPtr[i].x = xyzGpuPtr[i].x;
        pcGpuPtr[i].y = xyzGpuPtr[i].y;
        pcGpuPtr[i].z = xyzGpuPtr[i].z;
        pcGpuPtr[i].b = bgraGpuPtr[i].r;
        pcGpuPtr[i].g = bgraGpuPtr[i].g;
        pcGpuPtr[i].r = bgraGpuPtr[i].b;
        pcGpuPtr[i].a = bgraGpuPtr[i].a;
    }

    /**
     * Fills a PointCloud2 message residing on the CPU from two GPU buffers (one for XYZ and one for BGRA).
     *
     * @param xyzGpu    XYZ buffer on the GPU
     * @param bgraGpu   BGRA buffer on the GPU
     * @param pcGpu     Point cloud buffer on the GPU (@see Point)
     * @param msg       Point cloud message with buffer on the CPU
     */
    void fillPointCloudMessageFromGpu(sl::Mat& xyzGpu, sl::Mat& bgraGpu, PointCloudGpu& pcGpu, sensor_msgs::PointCloud2Ptr const& msg) {
        assert(bgraGpu.getWidth() >= xyzGpu.getWidth());
        assert(bgraGpu.getHeight() >= xyzGpu.getHeight());
        assert(bgraGpu.getChannels() == 4);
        assert(xyzGpu.getChannels() == 3);
        assert(msg);

        auto* bgraGpuPtr = bgraGpu.getPtr<sl::uchar4>(sl::MEM::GPU);
        auto* xyzGpuPtr = xyzGpu.getPtr<sl::float4>(sl::MEM::GPU);
        msg->is_bigendian = __BYTE_ORDER__ == __ORDER_BIG_ENDIAN__;
        msg->is_dense = false;  
        msg->height = bgraGpu.getHeight();
        msg->width = bgraGpu.getWidth();
        fillPointCloudMessageHeader(msg);
        size_t size = msg->width * msg->height;

        pcGpu.resize(size);
        Point* pcGpuPtr = pcGpu.data().get();
        dim3 threadsPerBlock{BLOCK_SIZE};
        dim3 numBlocks{static_cast<uint>(std::ceil(static_cast<float>(size) / BLOCK_SIZE))};
        fillPointCloudMessageKernel<<<numBlocks, threadsPerBlock>>>(xyzGpuPtr, bgraGpuPtr, pcGpuPtr, size);
        checkCudaError(hipPeekAtLastError());
        checkCudaError(hipMemcpy(msg->data.data(), pcGpuPtr, size * sizeof(Point), hipMemcpyDeviceToHost));
    }

    void checkCudaError(hipError_t err) {
        if (err == hipSuccess) return;

        ROS_ERROR_STREAM("CUDA error: " << hipGetErrorString(err));
        throw std::runtime_error("CUDA error");
    }

} // namespace mrover
