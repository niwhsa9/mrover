#include "hip/hip_runtime.h"
#include <NvInfer.h>
#include <NvInferRuntime.h>
#include <NvInferRuntimeBase.h>
#include <NvOnnxParser.h>
#include <hip/hip_runtime_api.h>

#include "ioHelper.cuh"

#include <memory>
#include <opencv4/opencv2/core/mat.hpp>
#include <opencv4/opencv2/core/types.hpp>
#include <opencv4/opencv2/imgproc.hpp>
#include <ostream>
#include <string>

#include "inference.cuh"

#include <array>
#include <filesystem>
#include <string_view>
#include <vector>

using namespace nvinfer1;

/**
* Example Code: @link https://github.com/NVIDIA-developer-blog/code-samples/blob/master/posts/TensorRT-introduction/simpleOnnx_1.cpp
* IExecutionContest @link https://docs.nvidia.com/deeplearning/tensorrt/api/c_api/classnvinfer1_1_1_i_execution_context.html
* ------------------------------------------------------
* For additional context see @link https://www.edge-ai-vision.com/2020/04/speeding-up-deep-learning-inference-using-tensorrt/
*/


/**
* cudaMemcpys CPU memory in inputTensor to GPU based on bindings
* Queues that tensor to be passed through model
* cudaMemcpys the result back to CPU memory
* Requires bindings, inputTensor, stream
* Modifies stream, outputTensor
*/
namespace mrover {

    //Constructor
    //  : logger{}, inputTensor{}, outputTensor{}, referenceTensor{}, stream{}
    Inference::Inference(std::string const& onnxModelPath, cv::Size modelInputShape = {640, 640}, std::string const& classesTxtFile = "")
        : mModelInputShape{modelInputShape} {

        mEngine = std::unique_ptr<ICudaEngine, Destroy<ICudaEngine>>{createCudaEngine(onnxModelPath)};
        if (!mEngine) throw std::runtime_error("Failed to create CUDA engine");

        mLogger.log(ILogger::Severity::kINFO, "Created CUDA Engine");

        // TODO: these are deprecated
        assert(mEngine->getNbBindings() == 2);
        assert(mEngine->bindingIsInput(0) ^ mEngine->bindingIsInput(1));

        mStream.emplace();

        mLogger.log(ILogger::Severity::kINFO, "Created CUDA stream");

        prepTensors();

        setUpContext();
    }

    // Initializes enginePtr with built engine
    ICudaEngine* Inference::createCudaEngine(std::string const& onnxModelPath) {
        // See link sfor additional context
        constexpr auto explicitBatch = 1U << static_cast<uint32_t>(NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);

        std::unique_ptr<IBuilder, Destroy<IBuilder>> builder{createInferBuilder(mLogger)};
        if (!builder) throw std::runtime_error("Failed to create Infer Builder");
        mLogger.log(ILogger::Severity::kINFO, "Created Infer Builder");

        std::unique_ptr<INetworkDefinition, Destroy<INetworkDefinition>> network{builder->createNetworkV2(explicitBatch)};
        if (!network) throw std::runtime_error("Failed to create Network Definition");
        mLogger.log(ILogger::Severity::kINFO, "Created Network Definition");

        std::unique_ptr<nvonnxparser::IParser, Destroy<nvonnxparser::IParser>> parser{nvonnxparser::createParser(*network, mLogger)};
        if (!parser) throw std::runtime_error("Failed to create ONNX Parser");
        mLogger.log(ILogger::Severity::kINFO, "Created ONNX Parser");

        std::unique_ptr<IBuilderConfig, Destroy<IBuilderConfig>> config{builder->createBuilderConfig()};
        if (!config) throw std::runtime_error("Failed to create Builder Config");
        mLogger.log(ILogger::Severity::kINFO, "Created Builder Config");

        // TODO: Not needed if we already have the engine file
        //Parse the onnx from file
        if (!parser->parseFromFile(onnxModelPath.c_str(), static_cast<int>(ILogger::Severity::kINFO))) {
            throw std::runtime_error("Failed to parse ONNX file");
        }

        config->setMemoryPoolLimit(MemoryPoolType::kWORKSPACE, 1 << 30);

        // auto profile = builder->createOptimizationProfile();
        // profile->setDimensions(network->getInput(0)->getName(), OptProfileSelector::kMIN, Dims4{1, 3, 256, 256});
        // profile->setDimensions(network->getInput(0)->getName(), OptProfileSelector::kOPT, Dims4{1, 3, 256, 256});
        // profile->setDimensions(network->getInput(0)->getName(), OptProfileSelector::kMAX, Dims4{32, 3, 256, 256});

        // config->addOptimizationProfile(profile);

        //Create runtime engine
        IRuntime* runtime = createInferRuntime(mLogger);

        std::filesystem::path enginePath("./tensorrt-engine.engine");

        //Check if engine file exists
        if (exists(enginePath)) {
            // TODO: error checking
            //Load engine from file
            std::ifstream inputFileStream("./tensorrt-engine.engine", std::ios::binary);
            std::stringstream engineBuffer;

            engineBuffer << inputFileStream.rdbuf();
            std::string enginePlan = engineBuffer.str();
            // TODO: deprecated
            return runtime->deserializeCudaEngine(enginePlan.data(), enginePlan.size(), nullptr);
        } else {
            IHostMemory* serializedEngine = builder->buildSerializedNetwork(*network, *config);
            if (!serializedEngine) throw std::runtime_error("Failed to serialize engine");

            //Create temporary engine for serializing
            ICudaEngine* tempEng = runtime->deserializeCudaEngine(serializedEngine->data(), serializedEngine->size());
            if (!tempEng) throw std::runtime_error("Failed to create temporary engine");

            //Save Engine to File
            auto trtModelStream = tempEng->serialize();
            std::ofstream outputFileStream("./tensorrt-engine.engine", std::ios::binary);
            outputFileStream.write(static_cast<const char*>(trtModelStream->data()), trtModelStream->size());
            outputFileStream.close();

            return tempEng;
        }
    }

    void Inference::setUpContext() {
        // Create Execution Context.
        mContext.reset(mEngine->createExecutionContext());

        Dims dims_i{mEngine->getBindingDimensions(0)};
        Dims4 inputDims{BATCH_SIZE, dims_i.d[1], dims_i.d[2], dims_i.d[3]};
        mContext->setBindingDimensions(0, inputDims);
    }

    void Inference::doDetections(cv::Mat& img) {
        //Do the forward pass on the network
        ROS_INFO("HI");
        launchInference(img.data, mOutputTensor.data);
        std::cout << *(mOutputTensor.data) << std::endl;
        //return Parser(outputTensor).parseTensor();
    }

    void Inference::launchInference(void* input, void* output) {
        assert(input);
        assert(output);
        assert(mContext);
        assert(mStream);

        int inputId = getBindingInputIndex(mContext.get());

        //Copy data to GPU memory
        std::cout << input << std::endl;
        std::cout << "ptr " << mBindings[inputId] << " size " << mInputDimensions.d[0] * mInputDimensions.d[1] * mInputDimensions.d[2] * sizeof(float) << std::endl;
        hipMemcpyAsync(mBindings[inputId], input, mInputDimensions.d[0] * mInputDimensions.d[1] * mInputDimensions.d[2] * sizeof(float), hipMemcpyHostToDevice, mStream.value());

        //Queue the async engine process
        mContext->enqueueV3(mStream.value());

        //Copy data to CPU memory
        hipMemcpyAsync(output, mBindings[1 - inputId], mOutputDimensions.d[0] * mOutputDimensions.d[1] * mOutputDimensions.d[2] * sizeof(float), hipMemcpyDeviceToHost, mStream.value());
    }


    /**
* Takes tensor bindings and allocates memory on the GPU for input and output tensors
* Requires enginePtr, bindings, inputTensor, and outputTensor
* Modifies bindings, inputTensor, and outputTensor
*/
    void Inference::prepTensors() {

        for (int i = 0; i < mEngine->getNbIOTensors(); i++) {
            const char* tensorName = mEngine->getIOTensorName(i);

            Dims dims{mEngine->getTensorShape(tensorName)};

            size_t size = accumulate(dims.d + 1, dims.d + dims.nbDims, BATCH_SIZE, std::multiplies<>());
            std::vector<int> sizes = {dims.d[1], dims.d[2], dims.d[3]};


            // Create CUDA buffer for Tensor.
            hipMalloc(&(mBindings)[i], BATCH_SIZE * size * sizeof(float));
        }

        mInputDimensions = Dims3(mModelInputShape.width, mModelInputShape.height, 3); //3 Is for the 3 RGB pixels
    }

    int Inference::getBindingInputIndex(IExecutionContext* context) {
        return context->getEngine().getTensorIOMode(context->getEngine().getIOTensorName(0)) != TensorIOMode::kINPUT; // 0 (false) if bindingIsInput(0), 1 (true) otherwise
    }
} // namespace mrover
