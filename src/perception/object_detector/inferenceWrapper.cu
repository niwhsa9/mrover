#include <NvInfer.h>
#include <NvInferRuntime.h>
#include <NvInferRuntimeBase.h>
#include <NvOnnxParser.h>
#include <hip/hip_runtime_api.h>

#include "ioHelper.cuh"

#include <memory>
#include <opencv4/opencv2/core/mat.hpp>
#include <opencv4/opencv2/core/types.hpp>
#include <string>

#include "inference.cuh"
#include "inferenceWrapper.hpp"

#include <array>
#include <string_view>
#include <vector>

using namespace nvinfer1;

/**
* Example Code: @link https://github.com/NVIDIA-developer-blog/code-samples/blob/master/posts/TensorRT-introduction/simpleOnnx_1.cpp
* IExecutionContest @link https://docs.nvidia.com/deeplearning/tensorrt/api/c_api/classnvinfer1_1_1_i_execution_context.html
* ------------------------------------------------------
* For additional context see @link https://www.edge-ai-vision.com/2020/04/speeding-up-deep-learning-inference-using-tensorrt/
*/


/**
* cudaMemcpys CPU memory in inputTensor to GPU based on bindings
* Queues that tensor to be passed through model
* cudaMemcpys the result back to CPU memory
* Requires bindings, inputTensor, stream
* Modifies stream, outputTensor
*/
namespace mrover {
    //Initialize the unique_ptr to the inference class
    InferenceWrapper::InferenceWrapper(std::string onnxModelPath, cv::Size modelInputShape = {640, 640}, std::string classesTxtFile = "") {
        inferencePtr.reset(new Inference(onnxModelPath, modelInputShape, classesTxtFile));
    }

    void InferenceWrapper::doDetections(cv::Mat& img) {
        inferencePtr->doDetections(img);
    }
} // namespace mrover
